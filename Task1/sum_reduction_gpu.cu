
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void sumRedKernel0(float* d_vec, unsigned int n)
{
    extern __shared__ float ds_partialSum[];
    unsigned int t = threadIdx.x;
    ds_partialSum[t] = d_vec[t];

    for (unsigned int stride = 1; stride < n; stride *= 2) {
        __syncthreads();
        if (t % (2 * stride) == 0 && t + stride < n)
            ds_partialSum[t] += ds_partialSum[t + stride];
    }

    if (t == 0)
        d_vec[0] = ds_partialSum[0];
}

__global__ void sumRedKernel1(float* d_vec, unsigned int n)
{
    extern __shared__ float ds_partialSum[];
    if (threadIdx.x < n)
        ds_partialSum[threadIdx.x] = d_vec[threadIdx.x];
    else
        ds_partialSum[threadIdx.x] = 0.0;

    for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (threadIdx.x < stride)
            ds_partialSum[threadIdx.x] += ds_partialSum[threadIdx.x + stride];
    }

    if (threadIdx.x == 0)
        d_vec[0] = ds_partialSum[0];
}

__global__ void sumRedKernel2(float* d_vec, unsigned int n)
{
    extern __shared__ float ds_partialSum[];
    unsigned int t = threadIdx.x;
    
    // Each thread loads two elements from global memory to shared memory
    if (t * 2 < n) {
        ds_partialSum[t] = d_vec[t * 2];
        if (t * 2 + 1 < n) {
            ds_partialSum[t] += d_vec[t * 2 + 1];
        }
    }

    // Parallel sum reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (t < stride) {
            ds_partialSum[t] += ds_partialSum[t + stride];
        }
    }

    // The 1st thread saves the sum from shared memory to global memory
    if (t == 0) {
        d_vec[0] = ds_partialSum[0];
    }
}

float sumRed(float* h_vec, unsigned int n, unsigned int version)
{
    unsigned int size = n * sizeof(float);
    float *d_vec;
    hipMalloc((void **) &d_vec, size);
    hipMemcpy(d_vec, h_vec, size, hipMemcpyHostToDevice);
    unsigned int blockSize, gridSize, sharedMemSize;
    switch (version) {
    case 0:
        gridSize = 1;
        blockSize = n;
        sharedMemSize = n * sizeof(float);
        sumRedKernel0<<<gridSize, blockSize, sharedMemSize>>>(d_vec, n);
        break;
    case 1:
        gridSize = 1;
        blockSize = 1 << ((int) ceil(log2(n)));
        sharedMemSize = blockSize * sizeof(float);
        sumRedKernel1<<<gridSize, blockSize, sharedMemSize>>>(d_vec, n);
        break;
    case 2:
    default:
        gridSize = 1;
        blockSize = (n + 1) / 2;
        sharedMemSize = blockSize * sizeof(float);
        sumRedKernel2<<<gridSize, blockSize, sharedMemSize>>>(d_vec, n);
        break;
    }
    float h_sum = 0.0;
    hipMemcpy(&h_sum, d_vec, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_vec);
    return h_sum;
}

int main(int argc, char *argv[])
{
    printf("Enter the number of elements to be summed up: ");
    unsigned int n;
    int ret;
    ret = scanf("%d", &n);
    if (ret != 1) {
        printf("Invalid input. Exiting.\n");
        return 1;
    }
   
    float *h_vec = (float *)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++)
        h_vec[i] = (float)i;

    printf("Enter the kernel case (0, 1, or 2): ");
    int kernel_case;
    ret = scanf("%d", &kernel_case);
    if (ret != 1) {
        printf("Invalid input. Exiting.\n");
        free(h_vec);
        return 1;
    }

    float sum = sumRed(h_vec, n, kernel_case);
    printf("sum = %.1f\n", sum);
    free(h_vec);
    return 0;
}
